#include "../include/AWcudaAllreduce.cuh"
#include <cstddef>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", __FILE__,   \
              __LINE__, hipGetErrorString(err));                              \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

float *read_data(const char *filename, size_t *dim) {
  FILE *file = fopen(filename, "r");
  if (!file) {
    perror("Err");
    return NULL;
  }

  size_t sz = 1000;
  *dim = 0;
  float *vec = (float *)malloc(sz * sizeof(float));
  if (!vec) {
    perror("mem allocation failed");
    fclose(file);
    return NULL;
  }

  char row[100];

  while (fgets(row, sizeof(row), file)) {
    if (*dim >= sz) {
      sz *= 2;
      float *temp = (float *)realloc(vec, sz * sizeof(float));
      if (!temp) {
        perror("mem allocation failed");
        free(vec);
        fclose(file);
        return NULL;
      }
      vec = temp;
    }
    vec[*dim] = strtof(row, NULL);
    (*dim)++;
  }

  fclose(file);
  return vec;
}

void write_dataf(const char *filename, float *data, size_t dim) {
  FILE *file = fopen(filename, "w");
  if (!file) {
    perror("Err");
    return;
  }

  for (size_t i = 0; i < dim; i++) {
    fprintf(file, "%f\n", data[i]);
  }

  fclose(file);
}
int main() {
  MPI_Init(NULL, NULL);
  size_t count;
  int rank, size;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &size);

  int deviceCount;
  CUDA_CHECK(hipSetDevice(0));

  float *h_sbuf;
  h_sbuf = read_data("smooth.in", &count);
  float *h_rbuf = (float *)malloc(count * sizeof(float));
  float *d_sbuf, *d_rbuf;
  hipMalloc((void **)&d_sbuf, count * sizeof(float));
  hipMemcpy(d_sbuf, h_sbuf, count * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **)&d_rbuf, count * sizeof(float));

  float eb = 0.0001;
  allreduce_ring_comprs_hom_sum(d_sbuf, d_rbuf, count, MPI_COMM_WORLD, eb);
  hipMemcpy(h_rbuf, d_rbuf, count * sizeof(float), hipMemcpyDeviceToHost);
  if (rank == 0) {
    write_dataf("smooth.out", h_rbuf, count);
  }
  hipFree(d_sbuf);
  hipFree(d_rbuf);
  MPI_Finalize();
}