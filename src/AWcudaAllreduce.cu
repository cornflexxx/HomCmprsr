#include "../include/GSZ.h"
#include "../include/GSZ_entry.h"
#include "../include/comprs_test.cuh"
#include <cstddef>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <stdio.h>

/* TODO : cuda Stream to overlap compression and quant prediction*/

#define MPI_call_check(call)                                                   \
  {                                                                            \
    int err_code = call;                                                       \
    if (err_code != MPI_SUCCESS) {                                             \
      char error_string[BUFSIZ];                                               \
      int length_of_error_string;                                              \
      MPI_Error_string(err_code, error_string, &length_of_error_string);       \
      fprintf(stderr, "\nMPI error in line %d : %s\n", __LINE__,               \
              error_string);                                                   \
      fflush(stderr);                                                          \
      MPI_Abort(MPI_COMM_WORLD, err_code);                                     \
    }                                                                          \
  }
#define CUDA_CHECK(call)                                                       \
  {                                                                            \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", __FILE__,   \
              __LINE__, hipGetErrorString(err));                              \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

#define COLL_BASE_COMPUTE_BLOCKCOUNT(COUNT, NUM_BLOCKS, SPLIT_INDEX,           \
                                     EARLY_BLOCK_COUNT, LATE_BLOCK_COUNT)      \
  EARLY_BLOCK_COUNT = LATE_BLOCK_COUNT = COUNT / NUM_BLOCKS;                   \
  SPLIT_INDEX = COUNT % NUM_BLOCKS;                                            \
  if (0 != SPLIT_INDEX) {                                                      \
    EARLY_BLOCK_COUNT = EARLY_BLOCK_COUNT + 1;                                 \
  }

int allreduce_ring_comprs_hom_sum(const float *d_sbuf, float *d_rbuf,
                                  size_t count, MPI_Comm comm, float eb) {
  int ret, line, rank, size, k, recv_from, send_to, block_count, inbi;
  int bsize, gsize;
  unsigned char *d_cmpReduceBytes;
  float *d_rtmpbuf;

  int *d_quant_predData;
  int early_segcount, late_segcount, split_rank, max_segcount;
  unsigned char *d_inbuf[2];
  ptrdiff_t block_offset, max_real_segsize;
  MPI_Request reqs[2] = {MPI_REQUEST_NULL, MPI_REQUEST_NULL};

  MPI_Comm_rank(comm, &rank); // get rank
  MPI_Comm_size(comm, &size); // get size of comm

  MPI_Status status;
  int count_;

  if (1 == size) {
    return MPI_SUCCESS;
  }

  size_t pad_nbEle = (count + 32768 - 1) / 32768 * 32768;
  COLL_BASE_COMPUTE_BLOCKCOUNT(pad_nbEle, size, split_rank, early_segcount,
                               late_segcount);
  early_segcount = (early_segcount % 4 == 0)
                       ? early_segcount
                       : early_segcount + (4 - early_segcount % 4);
  late_segcount = (late_segcount % 4 == 0)
                      ? late_segcount
                      : late_segcount + (4 - late_segcount % 4);

  max_segcount = early_segcount;
  max_real_segsize = max_segcount * sizeof(float);

  size_t padded_count =
      early_segcount * split_rank + late_segcount * (size - split_rank);
  CUDA_CHECK(hipMalloc((void **)&d_rtmpbuf, padded_count * sizeof(float)));
  CUDA_CHECK(
      hipMalloc((void **)&d_quant_predData, max_segcount * sizeof(int)));
  CUDA_CHECK(
      hipMalloc((void **)&d_cmpReduceBytes, max_segcount * sizeof(float)));

  hipMalloc((void **)&d_inbuf[0], max_real_segsize);
  if (size > 2) {
    hipMalloc((void **)&d_inbuf[1], max_real_segsize);
  }

  CUDA_CHECK(hipMemcpy(d_rtmpbuf, d_sbuf, count * sizeof(float),
                        hipMemcpyDeviceToDevice));

  send_to = (rank + 1) % size;
  recv_from = (rank + size - 1) % size;
  size_t cmpSize;
  inbi = 0;
  block_offset = ((rank < split_rank)
                      ? ((ptrdiff_t)rank * (ptrdiff_t)early_segcount)
                      : (ptrdiff_t)split_rank * early_segcount +
                            ((ptrdiff_t)rank - split_rank) * late_segcount);
  block_count = ((rank < split_rank) ? early_segcount : late_segcount);

  GSZ_compress_deviceptr_outlier(d_rtmpbuf + block_offset, d_cmpReduceBytes,
                                 block_count, &cmpSize, eb);
  CUDA_CHECK(hipGetLastError());
  MPI_call_check(MPI_Irecv(d_inbuf[inbi], max_real_segsize, MPI_BYTE, recv_from,
                           0, comm, &reqs[inbi]));
  MPI_call_check(MPI_Send(d_cmpReduceBytes, cmpSize + (cmpSize * 0.1), MPI_BYTE,
                          send_to, 0, comm));
  for (k = 2; k < size; k++) {
    const int prevblock = (rank + size - k + 1) % size;
    inbi = inbi ^ 0x1;
    block_offset = ((rank < split_rank)
                        ? ((ptrdiff_t)rank * (ptrdiff_t)early_segcount)
                        : (ptrdiff_t)split_rank * early_segcount +
                              ((ptrdiff_t)rank - split_rank) * late_segcount);
    block_count = ((rank < split_rank) ? early_segcount : late_segcount);
    bsize = dec_tblock_size;
    gsize = (block_count + bsize * dec_chunk - 1) / (bsize * dec_chunk);
    dim3 grid(gsize);
    dim3 block(bsize);
    kernel_quant_prediction<<<grid, block>>>(d_rtmpbuf + block_offset,
                                             d_quant_predData, eb, block_count);
    CUDA_CHECK(hipGetLastError());

    MPI_call_check(MPI_Irecv(d_inbuf[inbi], max_real_segsize, MPI_BYTE,
                             recv_from, 0, comm, &reqs[inbi]));

    MPI_call_check(MPI_Wait(&reqs[inbi ^ 0x1], &status));

    homomorphic_sum(d_inbuf[inbi ^ 0x1], d_quant_predData, d_cmpReduceBytes,
                    block_count, eb, &cmpSize);
    CUDA_CHECK(hipGetLastError());

    MPI_call_check(MPI_Send(d_cmpReduceBytes, cmpSize + (cmpSize * 0.1),
                            MPI_BYTE, send_to, 0, comm));
  }
  MPI_call_check(MPI_Wait(&reqs[inbi], &status));
  recv_from = (rank + 1) % size;
  block_offset = ((rank < split_rank)
                      ? ((ptrdiff_t)rank * (ptrdiff_t)early_segcount)
                      : (ptrdiff_t)split_rank * early_segcount +
                            ((ptrdiff_t)rank - split_rank) * late_segcount);
  block_count = ((rank < split_rank) ? early_segcount : late_segcount);
  bsize = dec_tblock_size;
  gsize = (block_count + bsize * dec_chunk - 1) / (bsize * dec_chunk);
  dim3 grid(gsize);
  dim3 block(bsize);

  kernel_quant_prediction<<<grid, block>>>(d_rtmpbuf + block_offset,
                                           d_quant_predData, eb, block_count);
  CUDA_CHECK(hipGetLastError());

  homomorphic_sum(d_inbuf[inbi], d_quant_predData, d_cmpReduceBytes,
                  block_count, eb, &cmpSize);
  CUDA_CHECK(hipGetLastError());
  GSZ_decompress_deviceptr_outlier(d_rtmpbuf + block_offset, d_cmpReduceBytes,
                                   block_count, cmpSize, eb);
  CUDA_CHECK(hipGetLastError());
  cmpSize = cmpSize + (cmpSize * 0.1);
  send_to = (rank + 1) % size;
  recv_from = (rank + size - 1) % size;
  for (k = 0; k < size - 1; k++) {
    const int recv_data_from = (rank + size - k) % size;
    const ptrdiff_t recv_block_offset =
        ((recv_data_from < split_rank)
             ? ((ptrdiff_t)recv_data_from * early_segcount)
             : (ptrdiff_t)split_rank * early_segcount +
                   ((ptrdiff_t)recv_data_from - split_rank) * late_segcount);
    block_count =
        ((recv_data_from < split_rank) ? early_segcount : late_segcount);

    MPI_call_check(MPI_Sendrecv(d_cmpReduceBytes, cmpSize, MPI_BYTE, send_to, 0,
                                d_inbuf[inbi], max_real_segsize, MPI_BYTE,
                                recv_from, 0, comm, &status));

    MPI_Get_count(&status, MPI_BYTE, &count_);
    cmpSize = count_;
    GSZ_decompress_deviceptr_outlier(d_rtmpbuf + recv_block_offset,
                                     d_inbuf[inbi], (size_t)block_count,
                                     cmpSize, eb);
    hipMemcpy(d_cmpReduceBytes, d_inbuf[inbi], cmpSize,
               hipMemcpyDeviceToDevice);
  }
  hipMemcpy(d_rbuf, d_rtmpbuf, count * sizeof(float),
             hipMemcpyDeviceToDevice);

  return 0;
}